#include "hip/hip_runtime.h"
// This is the REAL "hello world" for CUDA!
// It takes the string "Hello ", prints it, then passes it to CUDA with an array
// of offsets. Then the offsets are added in parallel to produce the string "World!"
// By Ingemar Ragnemalm 2010

#include <vector>
#include <iostream>
#include <stdio.h>
#include <helper_functions.h>
#include <hip/hip_vector_types.h>
#include <opencv2/opencv.hpp>

//////////////////////////////////////////////////
// Utils
//////////////////////////////////////////////////

#define PI2 1.5707963f
#define PI  3.1415927f
#define TAU 6.2831853f

#define GRID_NUM 32

__device__
float myLerp(float a, float b, float k)
{
    //return unsigned char(a * (1.f - k) + b * k);
    return a * (1.f - k) + b * k;
}

__device__
float myClamp(float value, float min, float max)
{
    if      (value < min) return min;
    else if (value > max) return max;
    else                  return value;
}

__device__
void normalize(float *v)
{
    const float k = 1.f / sqrtf(v[0] * v[0] + v[1] * v[1] + v[2] * v[2]);
    v[0] *= k;
    v[1] *= k;
    v[2] *= k;
}


//////////////////////////////////////////////////
// _to_img
//////////////////////////////////////////////////

__device__
void rect_to_img(int h, int w, const float* v, float& i, float& j)
{
    i = h * (       acosf (v[1])        / PI);
    j = w * (0.5f + atan2f(v[0], -v[2]) / TAU);
}

//////////////////////////////////////////////////
// _to_env
//////////////////////////////////////////////////

__device__
void cube_to_env(unsigned f, float i, float j, int h, int w, float* v)
{
    const int p[6][3][3] = {
        {{  0,  0, -1 }, {  0, -1,  0 }, {  1,  0,  0 }},
        {{  0,  0,  1 }, {  0, -1,  0 }, { -1,  0,  0 }},
        {{  1,  0,  0 }, {  0,  0,  1 }, {  0,  1,  0 }},
        {{  1,  0,  0 }, {  0,  0, -1 }, {  0, -1,  0 }},
        {{  1,  0,  0 }, {  0, -1,  0 }, {  0,  0,  1 }},
        {{ -1,  0,  0 }, {  0, -1,  0 }, {  0,  0, -1 }},
    };

    const float y = 2.0f * i / h - 1.0f;
    const float x = 2.0f * j / w - 1.0f;

    v[0] = p[f][0][0] * x + p[f][1][0] * y + p[f][2][0];
    v[1] = p[f][0][1] * x + p[f][1][1] * y + p[f][2][1];
    v[2] = p[f][0][2] * x + p[f][1][2] * y + p[f][2][2];

    normalize(v);
}

//////////////////////////////////////////////////
// Filters
//////////////////////////////////////////////////

__device__
void filter_linear(const unsigned char* srcImg, int srcHeight, int srcWidth, float i, float j, float* p)
{
    const float ii = myClamp(i - 0.5f, 0.0f, srcHeight - 1.0f);
    const float jj = myClamp(j - 0.5f, 0.0f, srcWidth - 1.0f);

    const long  i0 = lrintf(floorf(ii)), i1 = lrintf(ceilf(ii));
    const long  j0 = lrintf(floorf(jj)), j1 = lrintf(ceilf(jj));

    const float di = ii - i0;
    const float dj = jj - j0;

    for(int k=0; k<3; k++) {
        p[k] += myLerp(
            myLerp(srcImg[i0*srcWidth*3 + j0*3 + k], srcImg[i0*srcWidth*3 + j1*3 + k], dj),
            myLerp(srcImg[i1*srcWidth*3 + j0*3 + k], srcImg[i1*srcWidth*3 + j1*3 + k], dj),
            di
            );
    }
}

//////////////////////////////////////////////////
// supersample
//////////////////////////////////////////////////

__global__
void cudaSupersample(
    const unsigned char *srcImg,
    unsigned char *dstImg,
    unsigned srcHeight,
    unsigned srcWidth,
    unsigned dstSize,
    int f
    //int f, int i, int j
)
{
    /*
    //dstImg[threadIdx.x * dstSize + threadIdx.y] = threadIdx.x * dstSize + threadIdx.y;
    dstImg[threadIdx.x * dstSize * 3 + threadIdx.y * 3 + 0] = 66;
    dstImg[threadIdx.x * dstSize * 3 + threadIdx.y * 3 + 1] = 66;
    dstImg[threadIdx.x * dstSize * 3 + threadIdx.y * 3 + 2] = 66;
    */
    int i = GRID_NUM*blockIdx.x + threadIdx.x;
    int j = GRID_NUM*blockIdx.y + threadIdx.y;
    //int f = 0;

    // Pattern
    const int rgss_pattern_size = 4;
    const float rgss_pattern[4][2] = {
        { 0.125f, 0.625f },
        { 0.375f, 0.125f },
        { 0.625f, 0.875f },
        { 0.875f, 0.375f }
    };

    float p[3] = { 0.0f, 0.0f, 0.0f };
    float I, J;

    for(int k=0 ; k<rgss_pattern_size ; k++) {
        const float ii = rgss_pattern[k][0] + i;
        const float jj = rgss_pattern[k][1] + j;

        float v[3] = { 0.0f, 0.0f, 0.0f };
        cube_to_env(f, ii, jj, dstSize, dstSize, v);
        rect_to_img(srcHeight, srcWidth, v, I, J);
        filter_linear(srcImg, srcHeight, srcWidth, I, J, p);
    }
    for(int k=0 ; k<3 ; k++) {
        dstImg[i*dstSize*3 + j*3 + k] = p[k] / rgss_pattern_size;
        //dstImg[i*dstSize*3 + j*3 + k] = 128; // TODO: !?
    }
}

void cudaProcess(
    const unsigned char *srcImg,
    unsigned char *dstImgs[6],
    unsigned srcHeight,
    unsigned srcWidth,
    unsigned dstSize
)
{
    static unsigned char *gpuSrcImg = NULL;
    static unsigned char *gpuDstImgs[6] = {NULL, NULL, NULL, NULL, NULL, NULL};

    //clock_t begin_time = clock(); // TODO: time
    if(!gpuSrcImg) {
        hipMalloc( (void**)&gpuSrcImg, srcHeight * srcWidth * 3 * sizeof(unsigned char)); // 3: # channels
        for(int f=0 ; f<6 ; f++)
            hipMalloc( (void**)&gpuDstImgs[f], dstSize * dstSize * 3 * sizeof(unsigned char)); // 3: # channels
    }
    //clock_t end_time = clock(); // TODO: time
    //std::cout << "alloc: " << float( clock () - begin_time ) /  CLOCKS_PER_SEC << " sec" << std::endl; // TODO: time

    hipMemcpy( gpuSrcImg, srcImg, srcHeight * srcWidth * 3 * sizeof(unsigned char), hipMemcpyHostToDevice);

	dim3 dimBlock( dstSize/GRID_NUM, dstSize/GRID_NUM);
	dim3 dimGrid( GRID_NUM, GRID_NUM );
    for(int f=0 ; f<6 ; f++) {
        cudaSupersample<<<dimGrid, dimBlock>>>(gpuSrcImg, gpuDstImgs[f], srcHeight, srcWidth, dstSize, f);
    }

    for(int f=0 ; f<6 ; f++) {
        int result = hipMemcpy( dstImgs[f], gpuDstImgs[f], dstSize * dstSize * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);
        //if(result == hipSuccess)
        //    printf("Copy success\n");
    }

	//hipFree( gpuSrcImg );
    //for(int f=0 ; f<6 ; f++)
    //    hipFree( gpuDstImgs[f] );
}

void sphere2cube(cv::Mat& src, std::vector<cv::Mat*>& dst, unsigned size)
{
    for(int i=0 ; i<6 ; i++) {
        dst.push_back(new cv::Mat(size, size, CV_8UC3, cv::Scalar(0, 0, 0)));
    }

    unsigned char *dstImgPtrs[6];
    dstImgPtrs[0] = dst[0]->data;
    dstImgPtrs[1] = dst[1]->data;
    dstImgPtrs[2] = dst[2]->data;
    dstImgPtrs[3] = dst[3]->data;
    dstImgPtrs[4] = dst[4]->data;
    dstImgPtrs[5] = dst[5]->data;
    
    cudaProcess(src.data, dstImgPtrs, src.rows, src.cols, size);
}

void test(int size)
{
    cv::Mat img = cv::imread("input/image_001.tiff", CV_LOAD_IMAGE_UNCHANGED);

    cv::Mat dstImgs0(size, size, CV_8UC3, cv::Scalar(0, 0, 0));
    cv::Mat dstImgs1(size, size, CV_8UC3, cv::Scalar(0, 0, 0));
    cv::Mat dstImgs2(size, size, CV_8UC3, cv::Scalar(0, 0, 0));
    cv::Mat dstImgs3(size, size, CV_8UC3, cv::Scalar(0, 0, 0));
    cv::Mat dstImgs4(size, size, CV_8UC3, cv::Scalar(0, 0, 0));
    cv::Mat dstImgs5(size, size, CV_8UC3, cv::Scalar(0, 0, 0));
    unsigned char *dstImgPtrs[6];
    dstImgPtrs[0] = dstImgs0.data;
    dstImgPtrs[1] = dstImgs1.data;
    dstImgPtrs[2] = dstImgs2.data;
    dstImgPtrs[3] = dstImgs3.data;
    dstImgPtrs[4] = dstImgs4.data;
    dstImgPtrs[5] = dstImgs5.data;

    clock_t begin_time = clock();
    cudaProcess(img.data, dstImgPtrs, img.rows, img.cols, size);
    clock_t end_time = clock();
    std::cout << float( clock () - begin_time ) /  CLOCKS_PER_SEC << std::endl;

    //imwrite("output/cuda_0.jpg", dstImgs0);
    //imwrite("output/cuda_1.jpg", dstImgs1);
    //imwrite("output/cuda_2.jpg", dstImgs2);
    //imwrite("output/cuda_3.jpg", dstImgs3);
    //imwrite("output/cuda_4.jpg", dstImgs4);
    //imwrite("output/cuda_5.jpg", dstImgs5);

    /*
    for(int i=0 ; i<size ; i++) {
        for(int j=0 ; j<size ; j++) {
            printf("(%d, ", dstImgPtrs[0][i*size*3+j*3+0]);
            printf("%d, ", dstImgPtrs[0][i*size*3+j*3+1]);
            printf("%d) ", dstImgPtrs[0][i*size*3+j*3+2]);
        }
        printf("...\n");
        //printf("\n");
    }
    printf(".\n");
    printf(".\n");
    printf(".\n");
    */
}

void test_sphere2cube(unsigned size)
{
    std::cout << "test_sphere2cube(" << size << ")" << std::endl;
    cv::Mat img = cv::imread("input/image_001.tiff", CV_LOAD_IMAGE_UNCHANGED);
    std::vector<cv::Mat*> dst;

    sphere2cube(img, dst, size);

    imwrite("output/cuda_cube_0.jpg", *(dst[0]));
    imwrite("output/cuda_cube_1.jpg", *(dst[1]));
    imwrite("output/cuda_cube_2.jpg", *(dst[2]));
    imwrite("output/cuda_cube_3.jpg", *(dst[3]));
    imwrite("output/cuda_cube_4.jpg", *(dst[4]));
    imwrite("output/cuda_cube_5.jpg", *(dst[5]));
}

/*
int main()
{
    //for(int i=0 ; i<24 ; i++) {
        //test(32*GRID_NUM);
        //test(32);
        //test(256);
        test_sphere2cube(1024);
    //}
    return 0;
}
*/
